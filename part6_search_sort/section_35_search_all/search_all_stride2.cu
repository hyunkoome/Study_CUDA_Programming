#include "hip/hip_runtime.h"
#include "./common.cpp"
#include <limits.h>

// input parameters
unsigned num = 16 * 1024 * 1024; // maximum num of inputs
unsigned bound = 1000 * 1000; // numbers will be ranged in [0..bound)
unsigned BLOCK_SIZE = 1024; // block size
unsigned stride = 512 * 1024; // stride between threads

// CUDA kernel function
__global__ void kernelSearch(const unsigned* pData, unsigned num, unsigned target, unsigned* pIndex, unsigned stride) {
	unsigned index = blockIdx.x * blockDim.x + threadIdx.x; // CUDA-provided index
	for (unsigned j = index; j < num; j += stride) {
		if (pData[j] == target) {
			pIndex[index] = j;
			index++;
		}
	}
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		num = procArg( argv[0], argv[1], 1024 );
		break;
	case 3:
		num = procArg( argv[0], argv[1], 1024 );
		BLOCK_SIZE = procArg( argv[0], argv[2], 32, 1024 );
		break;
	case 4:
		num = procArg( argv[0], argv[1], 1024 );
		BLOCK_SIZE = procArg( argv[0], argv[2], 32, 1024 );
		stride = procArg( argv[0], argv[3], 32 );
		break;
	default:
		printf("usage: %s [num] [blocksize] [stride]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	printf("num = %u, BLOCK_SIZE = %u, stride = %u\n", num, BLOCK_SIZE, stride);
	printf("bound = %u\n", bound);
	// host-side data
	unsigned* vecData = nullptr;
	unsigned* vecIndex = nullptr;
	try {
		vecData = new unsigned[num];
		vecIndex = new unsigned[stride];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data to A and B
	srand( 0 );
	setRandomData<unsigned>( vecData, num, bound );
	unsigned targetValue = vecData[num - 1];
	printf("targetValue = %u\n", targetValue);
	memset( vecIndex, 0xFF, stride * sizeof(unsigned) );
	// device-side data
	unsigned* dev_vecData = nullptr;
	unsigned* dev_vecIndex = nullptr;
	hipMalloc( (void**)&dev_vecData, num * sizeof(unsigned) );
	hipMalloc( (void**)&dev_vecIndex, stride * sizeof(unsigned) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	ELAPSED_TIME_BEGIN(1);
	hipMemcpy( dev_vecData, vecData, num * sizeof(unsigned), hipMemcpyHostToDevice );
	hipMemcpy( dev_vecIndex, vecIndex, stride * sizeof(unsigned), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel call
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	dim3 dimGrid(div_up(stride, dimBlock.x), 1, 1);
	CUDA_PRINT_CONFIG( num );
	ELAPSED_TIME_BEGIN(0);
	kernelSearch <<< dimGrid, dimBlock>>>( dev_vecData, num, targetValue, dev_vecIndex, stride );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( vecIndex, dev_vecIndex, stride * sizeof(unsigned), hipMemcpyDeviceToHost );
	ELAPSED_TIME_END(1);
	CUDA_CHECK_ERROR();
	// free device memory
	hipFree( dev_vecData );
	hipFree( dev_vecIndex );
	CUDA_CHECK_ERROR();
	// check the result
	unsigned found = 0;
	for (unsigned i = 0; i < stride; ++i) {
		if (vecIndex[i] < num) {
			unsigned index = vecIndex[i];
			printf("vecData[%d]= %d\n", index, vecData[index]);
			found++;
		}
	}
	printf("%d locations are found\n", found);
	printVec( "vecIndex", vecIndex, found );
	printVec( "vecData", vecData, num );
	// cleaning
	delete[] vecData;
	delete[] vecIndex;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
