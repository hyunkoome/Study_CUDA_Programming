#include "hip/hip_runtime.h"
#include "./common.cpp"
#include <limits.h>
#include <algorithm>
#include <exception>
using namespace std;

// input parameters
unsigned num = 16 * 1024 * 1024; // maximum num of inputs
unsigned bound = 1000 * 1000; // numbers will be ranged in [0..bound)
unsigned BLOCK_SIZE = 1024; // block size
unsigned cnt = 1024; // each thread process this amount of input data

__device__ unsigned dev_index = UINT_MAX;

// CUDA kernel function
__global__ void kernelSearch(const unsigned* pData, unsigned num, unsigned target, unsigned count) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; // CUDA-provided index
	int first = min( i * count, num );
	int last = min( (i + 1) * count, num ) - 1;
	if (pData[first] <= target && target <= pData[last]) {
		while (first <= last) {
			register unsigned mid = (first + last) / 2;
			if (target == pData[mid]) {
				atomicMin( &dev_index, mid );
				last = first - 1;
			} else if (target < pData[mid]) {
				last = mid - 1;
			} else {
				first = mid + 1;
			}
		}
	}
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		num = procArg( argv[0], argv[1], 1024 );
		break;
	case 3:
		num = procArg( argv[0], argv[1], 1024 );
		BLOCK_SIZE = procArg( argv[0], argv[2], 32, 1024 );
		break;
	case 4:
		num = procArg( argv[0], argv[1], 1024 );
		BLOCK_SIZE = procArg( argv[0], argv[2], 32, 1024 );
		cnt = procArg( argv[0], argv[3], 32 );
		break;
	default:
		printf("usage: %s [num] [blocksize] [count]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	printf("num = %u, BLOCK_SIZE = %u, count = %u\n", num, BLOCK_SIZE, cnt);
	printf("bound = %u\n", bound);
	// host-side data
	unsigned* vecData = nullptr;
	try {
		vecData = new unsigned[num];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data to A and B
	srand( 0 );
	setRandomData( vecData, num, bound );
	unsigned targetValue = vecData[num - 1];
	printf("targetValue = %u\n", targetValue);
	// we need to sort it, for the binary search
	std::sort(vecData, vecData + num);
	// device-side data
	unsigned* dev_vecData = nullptr;
	hipMalloc( (void**)&dev_vecData, num * sizeof(unsigned) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	ELAPSED_TIME_BEGIN(1);
	hipMemcpy( dev_vecData, vecData, num * sizeof(unsigned), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel call
	dim3 dimBlock(BLOCK_SIZE, 1, 1); // only one block
	dim3 dimGrid(div_up(num, cnt * dimBlock.x), 1, 1);
	CUDA_PRINT_CONFIG( num );
	ELAPSED_TIME_BEGIN(0);
	kernelSearch <<< dimGrid, dimBlock>>>( dev_vecData, num, targetValue, cnt );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	unsigned index = UINT_MAX;
	hipMemcpyFromSymbol( &index, HIP_SYMBOL(dev_index), sizeof(unsigned), 0, hipMemcpyDeviceToHost );
	ELAPSED_TIME_END(1);
	CUDA_CHECK_ERROR();
	// free device memory
	hipFree( dev_vecData );
	CUDA_CHECK_ERROR();
	// check the result
	if (index >= num) {
		printf("NOT FOUND: target value '%u' not found\n", targetValue);
	} else {
		printf("FOUND: vecData[%d] = %d\n", index, vecData[index]);
	}
	printVec( "vecData", vecData, num );
	// cleaning
	delete[] vecData;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
