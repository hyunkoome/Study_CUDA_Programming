#include "hip/hip_runtime.h"
#include "./common.cpp"
#include <algorithm>

// input parameters
const unsigned BLOCK_SIZE = 1024;
const unsigned TOTAL_NUM = 16 * (2 * BLOCK_SIZE); // max total num data
const unsigned bound = 1000 * 1000; // numbers will be ranged in [0..bound)
enum {
	DECREASING = 0,
	INCREASING = 1,
};
unsigned direction = INCREASING;

int compLess( const void* lhs, const void* rhs ) {
	uint32_t lval = *(static_cast<const uint32_t*>(lhs));
	uint32_t rval = *(static_cast<const uint32_t*>(rhs));
	return (lval - rval);
}

int compGreater( const void* lhs, const void* rhs ) {
	uint32_t lval = *(static_cast<const uint32_t*>(lhs));
	uint32_t rval = *(static_cast<const uint32_t*>(rhs));
	return (rval - lval);
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		direction = procArg( argv[0], argv[1], 0, 1 );
		break;
	default:
		printf("usage: %s [direction] with 0=decreasing, 1=increasing\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	printf("BLOCK_SIZE = %d, TOTAL_NUM = %d\n", BLOCK_SIZE, TOTAL_NUM);
	printf("bound = %d, dir = %s\n", bound, (direction == 0) ? "DECREASING" : "INCREASING" );
	// host-side data
	uint32_t* src = nullptr; // original data
	uint32_t* dst = nullptr; // processed by CPU
	try {
		src = new uint32_t[TOTAL_NUM];
		dst = new uint32_t[TOTAL_NUM];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data
	srand( 0 );
	setRandomData( src, TOTAL_NUM, bound );
	// configuration for each block
	unsigned unit_size = (2 * BLOCK_SIZE);
	unsigned num_units = TOTAL_NUM / unit_size;
	printf("UNIT SIZE = %d\n", unit_size);
	printf("NUM UNITS = %d\n", num_units);
	// CPU processing
	memcpy( dst, src, TOTAL_NUM * sizeof(uint32_t) );
	ELAPSED_TIME_BEGIN(0);
	if (direction == INCREASING) {
		for (unsigned i = 0; i < num_units; ++i) {
			std::sort( dst + i * (2 * BLOCK_SIZE), dst + (i + 1) * (2 * BLOCK_SIZE) );
		}
	} else {
		for (unsigned i = 0; i < num_units; ++i) {
			std::sort( dst + i * (2 * BLOCK_SIZE), dst + (i + 1) * (2 * BLOCK_SIZE), std::greater<uint32_t>() );
		}
	}
	ELAPSED_TIME_END(0);
	// another processing with CPU
	if (direction == INCREASING) {
		for (unsigned i = 0; i < num_units; ++i) {
			qsort( src + i * (2 * BLOCK_SIZE), (2 * BLOCK_SIZE), sizeof(uint32_t), compLess );
		}
	} else {
		for (unsigned i = 0; i < num_units; ++i) {
			qsort( src + i * (2 * BLOCK_SIZE), (2 * BLOCK_SIZE), sizeof(uint32_t), compGreater );
		}
	}
	// show the result
	uint32_t err = getTotalDiff( src, dst, TOTAL_NUM );
	printf("total diff = %d\n", err);
	printf("%d sorted units:\n", num_units);
	for (unsigned i = 0; i < num_units; ++i) {
		printVec( "dst", dst + i * (2 * BLOCK_SIZE), (2 * BLOCK_SIZE) );
	}
	// cleaning
	delete[] src;
	delete[] dst;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
