#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
const unsigned BLOCK_SIZE = 1024;
unsigned NUM = 256 * 1024 * 1024; // num of samplings

// CUDA kernel function
__global__ void kernelHalfHalf( float* dst, const float* src, unsigned num, unsigned half ) {
	__shared__ float s_src[BLOCK_SIZE]; // source area
	__shared__ float s_dst[BLOCK_SIZE];
	register unsigned gx = blockIdx.x * blockDim.x + threadIdx.x;
	register unsigned tx = threadIdx.x;
	// read from global memory
	s_src[tx] = src[gx];
	__syncthreads(); // intentionally used
	// ... some action followed
	// main action
	if (tx < BLOCK_SIZE / 2) { // left half
		s_dst[tx] = s_src[2 * tx];
	} else {
		s_dst[tx] = s_src[2 * (tx - BLOCK_SIZE / 2) + 1];
	}
	__syncthreads();
	// write to the global memory
	register unsigned shift = blockIdx.x * blockDim.x / 2;
	if (tx < BLOCK_SIZE / 2) {
		dst[shift + tx] = s_dst[tx];
	} else {
		dst[half + shift + tx - BLOCK_SIZE / 2] = s_dst[tx];
	}
}

int main( const int argc, const char* argv[] ) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		NUM = procArg( argv[0], argv[1], 1 );
		break;
	default:
		printf("usage: %s [num]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	if (NUM % 2 != 0) {
		printf("%s: ERROR: invalid num = %d\n", argv[0], NUM);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
	}
	printf("num=%d\n", NUM);
	// host-side data
	float* src = new float[NUM];
	float* dst = new float[NUM];
	// set random data
	srand( 0 );
	setNormalizedRandomData( src, NUM );
	// device-side data
	float* dev_src = nullptr;
	float* dev_dst = nullptr;
	// allocate device memory
	hipMalloc( (void**)&dev_src, NUM * sizeof(float) );
	hipMalloc( (void**)&dev_dst, NUM * sizeof(float) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	ELAPSED_TIME_BEGIN(1);
	hipMemcpy( dev_src, src, NUM * sizeof(float), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	dim3 dimBlock( BLOCK_SIZE, 1, 1 );
	dim3 dimGrid( div_up(NUM, dimBlock.x), 1, 1 );
	CUDA_PRINT_CONFIG( NUM );
	ELAPSED_TIME_BEGIN(0);
	kernelHalfHalf <<< dimGrid, dimBlock>>>( dev_dst, dev_src, NUM, NUM / 2 );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( dst, dev_dst, NUM * sizeof(float), hipMemcpyDeviceToHost );
	ELAPSED_TIME_END(1);
	CUDA_CHECK_ERROR();
	// free device memory
	hipFree( dev_src );
	hipFree( dev_dst );
	CUDA_CHECK_ERROR();
	// check the result
	float sumDst = getSum( dst, NUM );
	float sumSrc = getSum( src, NUM );
	float diff = fabsf( sumDst - sumSrc );
	printf("sumDst = %f\n", sumDst);
	printf("sumSrc = %f\n", sumSrc);
	printf("diff(sumDst, sumSrc) = %f\n", diff);
	printVec( "dst", dst, NUM );
	printVec( "src", src, NUM );
	// cleaning
	delete[] src;
	delete[] dst;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
