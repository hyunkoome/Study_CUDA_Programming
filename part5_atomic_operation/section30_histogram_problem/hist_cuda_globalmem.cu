#include "hip/hip_runtime.h"
#include "./common.cpp"
#include "./image.cpp"

const unsigned image_width = 640;
const unsigned image_height = 400;
const unsigned HIST_SIZE = 32; // histogram levels

// CUDA kernel function
__global__ void kernelHist(const unsigned char* img, unsigned num, unsigned* hist) {
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x; // CUDA-provided index
	if (i < num) {
		unsigned pixelVal = (unsigned)(img[i]) / 8;
		atomicAdd( &(hist[pixelVal]), 1 );
	}
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	default:
		printf("usage: %s\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	// host-side data
	unsigned* vecHist = nullptr;
	try {
		vecHist = new unsigned[HIST_SIZE];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE); // ENOMEM: cannot allocate memory
	}
	// set data to be zero
	memset(vecHist, 0, HIST_SIZE * sizeof(unsigned));
	// device-side data
	unsigned char* dev_image = nullptr;
	unsigned* dev_vecHist = nullptr;
	hipMalloc( (void**)&dev_image, sizeof(grayscale_data) );
	hipMalloc( (void**)&dev_vecHist, HIST_SIZE * sizeof(unsigned) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	ELAPSED_TIME_BEGIN(1);
	hipMemcpy( dev_image, grayscale_data, sizeof(grayscale_data), hipMemcpyHostToDevice );
	hipMemcpy( dev_vecHist, vecHist, HIST_SIZE * sizeof(unsigned), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid(div_up(image_width * image_height, dimBlock.x), 1, 1);
	CUDA_PRINT_CONFIG( image_width * image_height );
	ELAPSED_TIME_BEGIN(0);
	kernelHist <<< dimGrid, dimBlock>>>( dev_image, image_width * image_height, dev_vecHist );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( vecHist, dev_vecHist, HIST_SIZE * sizeof(unsigned), hipMemcpyDeviceToHost );
	ELAPSED_TIME_END(1);
	CUDA_CHECK_ERROR();
	// free device memory
	hipFree( dev_image );
	hipFree( dev_vecHist );
	CUDA_CHECK_ERROR();
	// check the result
	printf("image pixels = %zu\n", sizeof(grayscale_data));
	printf("histogram levels = %u\n", HIST_SIZE);
	unsigned sum = 0;
	for (register unsigned i = 0; i < HIST_SIZE; ++i) {
		printf("hist[%2d] = %8u\n", i, vecHist[i]);
		sum += vecHist[i];
	}
	printf("sum = %u\n", sum);
	// cleaning
	delete[] vecHist;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
