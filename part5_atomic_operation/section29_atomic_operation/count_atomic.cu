#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
const unsigned BLOCK_SIZE = 1024;
unsigned NUM = 64 * 1024 * 1024; // num of samplings

// CUDA device variables
__device__ unsigned long long dev_count = 0;

// CUDA kernel function
__global__ void kernelCount( unsigned num ) {
	register unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < num) {
		atomicAdd( &dev_count, 1ULL );
	}
}

int main( const int argc, const char* argv[] ) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		NUM = procArg( argv[0], argv[1], 1 );
		break;
	default:
		printf("usage: %s [num]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	printf("NUM = %d\n", NUM);
	// CUDA kernel launch
	dim3 dimBlock( BLOCK_SIZE, 1, 1 );
	dim3 dimGrid( div_up(NUM, dimBlock.x), 1, 1 );
	ELAPSED_TIME_BEGIN(0);
	kernelCount <<< dimGrid, dimBlock>>>( NUM );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	unsigned long long count;
	hipMemcpyFromSymbol( &count, HIP_SYMBOL(dev_count), sizeof(unsigned long long), 0, hipMemcpyDeviceToHost );
	CUDA_CHECK_ERROR();
	// check the result
	printf("num thread launched = %d\n", NUM);
	printf("count = %llu\n", count);
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
