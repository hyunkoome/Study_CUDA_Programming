#include "hip/hip_runtime.h"
#include "./common.cpp"

const unsigned SIZE = 256 * 1024 * 1024; // big-size elements

// CUDA kernel function
__global__ void singleKernelVecAdd( float* c, const float* a, const float* b ) {
	for (register unsigned i = 0; i < SIZE; ++i) {
		c[i] = a[i] + b[i];
	}
}


int main(void) {
	// host-side data
	float* vecA = nullptr;
	float* vecB = nullptr;
	float* vecC = nullptr;
	try {
		vecA = new float[SIZE];
		vecB = new float[SIZE];
		vecC = new float[SIZE];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(1);
	}
	// set random data
	srand( 0 );
	setNormalizedRandomData( vecA, SIZE );
	setNormalizedRandomData( vecB, SIZE );
	// device-side data
	float* dev_vecA = nullptr;
	float* dev_vecB = nullptr;
	float* dev_vecC = nullptr;
	// allocate device memory
	hipMalloc( (void**)&dev_vecA, SIZE * sizeof(float) );
	hipMalloc( (void**)&dev_vecB, SIZE * sizeof(float) );
	hipMalloc( (void**)&dev_vecC, SIZE * sizeof(float) );
	// copy to device from host
	ELAPSED_TIME_BEGIN(1);
	hipMemcpy( dev_vecA, vecA, SIZE * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_vecB, vecB, SIZE * sizeof(float), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	ELAPSED_TIME_BEGIN(0);
	singleKernelVecAdd <<< 1, 1>>>( dev_vecC, dev_vecA, dev_vecB );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( vecC, dev_vecC, SIZE * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK_ERROR();
	ELAPSED_TIME_END(1);
	// free device memory
	hipFree( dev_vecA );
	hipFree( dev_vecB );
	hipFree( dev_vecC );
	CUDA_CHECK_ERROR();
	// check the result
	float sumA = getSum( vecA, SIZE );
	float sumB = getSum( vecB, SIZE );
	float sumC = getSum( vecC, SIZE );
	float diff = fabsf( sumC - (sumA + sumB) );
	printf("SIZE = %d\n", SIZE);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printf("sumC = %f\n", sumC);
	printf("diff(sumC, sumA+sumB) =  %f\n", diff);
	printf("diff(sumC, sumA+sumB) / SIZE =  %f\n", diff / SIZE);
	printVec( "vecA", vecA, SIZE );
	printVec( "vecB", vecB, SIZE );
	printVec( "vecC", vecC, SIZE );
	// cleaning
	delete[] vecA;
	delete[] vecB;
	delete[] vecC;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
