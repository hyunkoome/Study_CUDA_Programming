/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <chrono>
using namespace std;
using namespace std::chrono;

// dummy big job
void bigJob(void) {
	int count = 0;
	for (int i = 0; i < 10000; ++i) {
		for (int j = 0; j < 10000; ++j) {
			count++;
		}
	}
	printf("we got %d counts.\n", count);
}

int main(void) {
	system_clock::time_point chrono_begin = system_clock::now();
	// work
	bigJob();
	// work done
	system_clock::time_point chrono_end = system_clock::now();
	// calculation
	microseconds chrono_elapsed_usec = duration_cast<microseconds>(chrono_end - chrono_begin);
	printf("elapsed time = %ld usec\n", (long)chrono_elapsed_usec.count());
	// done
	return 0;
}

// usec = micro second = 1/1,000,000 sec
