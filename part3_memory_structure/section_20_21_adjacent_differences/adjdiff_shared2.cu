#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
unsigned num = 16 * 1024 * 1024; // num data
unsigned blocksize = 1024; // shared mem buf size

// CUDA kernel function
__global__ void kernelAdjDiff(float* b, const float* a, int num) {
	extern __shared__ float s_data[];  // area size not fixed at compile time
	register unsigned tx = threadIdx.x;
	register unsigned i = blockIdx.x * blockDim.x + threadIdx.x; // CUDA-provided index
	if (i < num) {
		s_data[tx] = a[i];
		__syncthreads();
		if (tx > 0) {
			b[i] = s_data[tx] - s_data[tx - 1];
		} else if (i > 0) {
			b[i] = s_data[tx] - a[i - 1];
		} else { // i == 0
			b[i] = s_data[tx] - 0.0f;
		}
	}
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		num = procArg( argv[0], argv[1], 1 );
		break;
	case 3:
		num = procArg( argv[0], argv[1], 1 );
		blocksize = procArg( argv[0], argv[2], 32, 1024 );
		break;
	default:
		printf("usage: %s [num] [blocksize]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	// host-side data
	float* vecA = nullptr;
	float* vecB = nullptr;
	try {
		vecA = new float[num];
		vecB = new float[num];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data
	srand( 0 );
	setNormalizedRandomData( vecA, num );
	// device-side data
	float* dev_vecA = nullptr;
	float* dev_vecB = nullptr;
	// allocate device memory
	ELAPSED_TIME_BEGIN(1);
	hipMalloc( (void**)&dev_vecA, num * sizeof(float) );
	hipMalloc( (void**)&dev_vecB, num * sizeof(float) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	hipMemcpy( dev_vecA, vecA, num * sizeof(float), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	dim3 dimBlock(blocksize, 1, 1);
	dim3 dimGrid(div_up(num, dimBlock.x), 1, 1);
	CUDA_PRINT_CONFIG( num );
	ELAPSED_TIME_BEGIN(0);
	kernelAdjDiff <<< dimGrid, dimBlock, blocksize * sizeof(float)>>>( dev_vecB, dev_vecA, num );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( vecB, dev_vecB, num * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK_ERROR();
	ELAPSED_TIME_END(1);
	// free device memory
	hipFree( dev_vecA );
	hipFree( dev_vecB );
	CUDA_CHECK_ERROR();
	// check the result
	float sumA = getSum( vecA, num );
	float sumB = getSum( vecB, num );
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printVec( "vecA", vecA, num );
	printVec( "vecB", vecB, num );
	// cleaning
	delete[] vecA;
	delete[] vecB;
	// done
	fflush(stdout);
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
