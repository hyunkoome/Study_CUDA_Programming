#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
unsigned num = 16 * 1024 * 1024; // num data

// CUDA kernel function
__global__ void kernelAdjDiff(float* b, const float* a, unsigned num) {
    // thread block 에서 1024개의 쓰레드를 동시에 돌릴 거라서, shared 메모리 크기도 1024로 세팅
    // 그래서 1024개가 동시에 읽어와서 실행되는 걸로 갈겁니다.
	__shared__ float s_data[1024];

	register unsigned tx = threadIdx.x; // local index
	register unsigned i = blockIdx.x * blockDim.x + threadIdx.x; // CUDA-provided index, global index
	if (i < num) {
		s_data[tx] = a[i]; // global memory -> shared memory 로 copy
		__syncthreads(); // race condition 때문에 동기화로 꼭 필요!!

        // shared memory -> global memory 로 copy, global b 변수로 결과를 넣어주니, b[i] = 로 사용됨 .
		if (tx > 0)
        {
			b[i] = s_data[tx] - s_data[tx - 1];
		}
        else if (i > 0) // tx == 0 인 경우, tx-1 의 값은 global 메모리 값인 a[i - 1]를 사용한다.
        {
			b[i] = s_data[tx] - a[i - 1];
		}
        else // i == 0 인 경우, i-1 은 값이 없으니깐, 0을 빼줌
        {
			b[i] = s_data[tx] - 0.0f;
		}
	}
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		num = procArg( argv[0], argv[1], 1 );
		break;
	default:
		printf("usage: %s [num]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	// host-side data
	float* vecA = nullptr;
	float* vecB = nullptr;
	try {
		vecA = new float[num];
		vecB = new float[num];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data
	srand( 0 );
	setNormalizedRandomData( vecA, num );
	// device-side data
	float* dev_vecA = nullptr;
	float* dev_vecB = nullptr;
	// allocate device memory
	ELAPSED_TIME_BEGIN(1);
	hipMalloc( (void**)&dev_vecA, num * sizeof(float) );
	hipMalloc( (void**)&dev_vecB, num * sizeof(float) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	hipMemcpy( dev_vecA, vecA, num * sizeof(float), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid(div_up(num, dimBlock.x), 1, 1);
	CUDA_PRINT_CONFIG( num );
	ELAPSED_TIME_BEGIN(0);
	kernelAdjDiff <<< dimGrid, dimBlock>>>( dev_vecB, dev_vecA, num );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( vecB, dev_vecB, num * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK_ERROR();
	ELAPSED_TIME_END(1);
	// free device memory
	hipFree( dev_vecA );
	hipFree( dev_vecB );
	CUDA_CHECK_ERROR();
	// check the result
	float sumA = getSum( vecA, num );
	float sumB = getSum( vecB, num );
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printVec( "vecA", vecA, num );
	printVec( "vecB", vecB, num );
	// cleaning
	delete[] vecA;
	delete[] vecB;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
