#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
unsigned num = 16 * 1024 * 1024; // num data

// CUDA kernel function
__global__ void kernelAdjDiff( float* b, const float* a, unsigned num ) {
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x; // CUDA-provided index
	if (i == 0) {
		b[i] = a[i] - 0.0f;
	} else if (i < num) {
		b[i] = a[i] - a[i - 1];
	}
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		num = procArg( argv[0], argv[1], 1 );
		break;
	default:
		printf("usage: %s [num]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	// host-side data
	float* vecA = nullptr;
	float* vecB = nullptr;
	try {
		vecA = new float[num];
		vecB = new float[num];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data
	srand( 0 );
	setNormalizedRandomData( vecA, num );
	// device-side data
	float* dev_vecA = nullptr;
	float* dev_vecB = nullptr;
	// allocate device memory
	ELAPSED_TIME_BEGIN(1);
	hipMalloc( (void**)&dev_vecA, num * sizeof(float) );
	hipMalloc( (void**)&dev_vecB, num * sizeof(float) );
	CUDA_CHECK_ERROR();
	// copy to device from host
	hipMemcpy( dev_vecA, vecA, num * sizeof(float), hipMemcpyHostToDevice );
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid(div_up(num, dimBlock.x), 1, 1);
	CUDA_PRINT_CONFIG( num );
	ELAPSED_TIME_BEGIN(0);
	kernelAdjDiff <<< dimGrid, dimBlock>>>( dev_vecB, dev_vecA, num );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy( vecB, dev_vecB, num * sizeof(float), hipMemcpyDeviceToHost );
	CUDA_CHECK_ERROR();
	ELAPSED_TIME_END(1);
	// free device memory
	hipFree( dev_vecA );
	hipFree( dev_vecB );
	CUDA_CHECK_ERROR();
	// check the result
	float sumA = getSum( vecA, num );
	float sumB = getSum( vecB, num );
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printVec( "vecA", vecA, num );
	printVec( "vecB", vecB, num );
	// cleaning
	delete[] vecA;
	delete[] vecB;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
