#include "hip/hip_runtime.h"
#include "./common.cpp"

// input parameters
const unsigned TILE_WIDTH = 32;
unsigned matsize = 1024; // num rows and also num cols

// CUDA kernel function: ASSUMPTION: (matsize % TILE_WIDTH == 0)
__global__ void kernelMatMul( float* C, const float* A, const float* B, unsigned matsize, size_t pitch_in_elem ) {
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];
	register unsigned ntiles = matsize / TILE_WIDTH; // A행렬의 경우 옆으로 가면서 읽을껀데, 타일단위로 끊을것임. 이때, 타일 개수 계산하기 위해서,
	register unsigned gy = blockIdx.y * blockDim.y + threadIdx.y; // y-coord, 저장할때 활용
	register unsigned gx = blockIdx.x * blockDim.x + threadIdx.x; // x-coord, 저장할때 활용
	register float sum = 0.0f;
	for (register unsigned tile = 0; tile < ntiles; ++tile) {
		register unsigned idxA = gy * pitch_in_elem + (tile * TILE_WIDTH + threadIdx.x);
		s_A[threadIdx.y][threadIdx.x] = A[idxA]; // A를 위한 타일, 옆으로 한줄 읽어옴(속도 높이기 위해)
		register unsigned idxB = (tile * TILE_WIDTH + threadIdx.y) * pitch_in_elem + gx;
		s_B[threadIdx.y][threadIdx.x] = B[idxB]; // B를 위한 타일, 옆으로 한줄 읽어옴(속도 높이기 위해)
		__syncthreads();
		for (register unsigned k = 0; k < TILE_WIDTH; ++k) {
			sum += s_A[threadIdx.y][k] * s_B[k][threadIdx.x]; // s_: 쉐어드 메모리에 대해서 곱하기를 함을 의미
		}
		__syncthreads();
	}
	register unsigned idxC = gy * pitch_in_elem + gx;
	C[idxC] = sum;
}

int main(const int argc, const char* argv[]) {
	// argv processing
	switch (argc) {
	case 1:
		break;
	case 2:
		matsize = procArg( argv[0], argv[1], 4 );
		break;
	default:
		printf("usage: %s [matsize]\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
		break;
	}
	if (matsize % 32 != 0) {
		printf("%s: only accepts multiples of 32\n", argv[0]);
		exit(EXIT_FAILURE); // EINVAL: invalid argument
	}
	// host-side data
	float* matA = nullptr;
	float* matB = nullptr;
	float* matC = nullptr;
	float* matD = nullptr;
	try {
		matA = new float[matsize * matsize];
		matB = new float[matsize * matsize];
		matC = new float[matsize * matsize];
		matD = new float[matsize * matsize];
	} catch (const exception& e) {
		printf("C++ EXCEPTION: %s\n", e.what());
		exit(EXIT_FAILURE);
	}
	// set random data
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize );
	setNormalizedRandomData( matB, matsize * matsize );
	// device-side data
	float* dev_matA = nullptr;
	float* dev_matB = nullptr;
	float* dev_matC = nullptr;
	// allocate device memory
	ELAPSED_TIME_BEGIN(1);
	size_t host_pitch = matsize * sizeof(float); // host side: packed compactly
	size_t dev_pitch = 0;
	hipMallocPitch( (void**)&dev_matA, &dev_pitch, matsize * sizeof(float), matsize );
	hipMallocPitch( (void**)&dev_matB, &dev_pitch, matsize * sizeof(float), matsize );
	hipMallocPitch( (void**)&dev_matC, &dev_pitch, matsize * sizeof(float), matsize );
	printf("dev_pitch = %zd byte, host_pitch = %zd byte\n", dev_pitch, host_pitch);
	CUDA_CHECK_ERROR();
	// copy to device from host
	hipMemcpy2D( dev_matA, dev_pitch, matA, host_pitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
	hipMemcpy2D( dev_matB, dev_pitch, matB, host_pitch, matsize * sizeof(float), matsize, hipMemcpyHostToDevice);
	CUDA_CHECK_ERROR();
	// CUDA kernel launch
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid(div_up(matsize, dimBlock.x), div_up(matsize, dimBlock.y), 1);
	assert(matsize % TILE_WIDTH == 0);
	assert(dev_pitch % sizeof(float) == 0);
	register unsigned pitch_in_elem = dev_pitch / sizeof(float);
	CUDA_PRINT_CONFIG_2D( matsize, matsize );
	ELAPSED_TIME_BEGIN(0);
	kernelMatMul <<< dimGrid, dimBlock>>>( dev_matC, dev_matA, dev_matB, matsize, pitch_in_elem );
	hipDeviceSynchronize();
	ELAPSED_TIME_END(0);
	CUDA_CHECK_ERROR();
	// copy to host from device
	hipMemcpy2D( matC, host_pitch, dev_matC, dev_pitch, matsize * sizeof(float), matsize, hipMemcpyDeviceToHost);
	ELAPSED_TIME_END(1);
	CUDA_CHECK_ERROR();
	// free device memory
	hipFree( dev_matA );
	hipFree( dev_matB );
	hipFree( dev_matC );
	CUDA_CHECK_ERROR();
	// check the result
	float sumA = getSum( matA, matsize * matsize );
	float sumB = getSum( matB, matsize * matsize );
	float sumC = getSum( matC, matsize * matsize );
	printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
	printf("sumA = %f\n", sumA);
	printf("sumB = %f\n", sumB);
	printf("sumC = %f\n", sumC);
	printMat( "matC", matC, matsize, matsize );
	printMat( "matA", matA, matsize, matsize );
	printMat( "matB", matB, matsize, matsize );
#if 0
	memset( matD, 0, matsize * matsize * sizeof(float) );
	for (register unsigned k = 0; k < matsize; ++k) {
		for (register unsigned r = 0; r < matsize; ++r) {
			for (register unsigned c = 0; c < matsize; ++c) {
				unsigned indC = r * matsize + c; // convert to 1D index
				unsigned indA = r * matsize + k; // convert to 1D index
				unsigned indB = k * matsize + c; // convert to 1D index
				matD[indC] += matA[indA] * matB[indB];
			}
		}
	}
	float sumD = getSum( matD, matsize * matsize );
	float rmsError = getRMS( matD, matC, matsize * matsize );
	printf("sumD = %f\n", sumD);
	printf("rmsErr(matC, matD) = %f\n", rmsError);
#endif
	// cleaning
	delete[] matA;
	delete[] matB;
	delete[] matC;
	delete[] matD;
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
