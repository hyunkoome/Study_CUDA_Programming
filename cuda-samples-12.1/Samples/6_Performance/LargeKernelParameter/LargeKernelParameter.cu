#include "hip/hip_runtime.h"
/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This is a simple test showing performance and usability
 * improvements with large kernel parameters introduced in CUDA 12.1
 */
#include <chrono>
#include <iostream>
#include <cassert>

// Utility includes
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace std::chrono;

#define TEST_ITERATIONS     (1000)
#define TOTAL_PARAMS        (8000)  // ints
#define KERNEL_PARAM_LIMIT  (1024)  // ints
#define CONST_COPIED_PARAMS (TOTAL_PARAMS - KERNEL_PARAM_LIMIT)

__constant__ int excess_params[CONST_COPIED_PARAMS];

typedef struct {
  int param[KERNEL_PARAM_LIMIT];
} param_t;

typedef struct {
  int param[TOTAL_PARAMS];
} param_large_t;

// Kernel with 4KB kernel parameter limit
__global__ void kernelDefault(__grid_constant__ const param_t p, int *result) {
  int tmp = 0;

  // accumulate kernel parameters
  for (int i = 0; i < KERNEL_PARAM_LIMIT; ++i) {
    tmp += p.param[i];
  }

  // accumulate excess values passed via const memory
  for (int i = 0; i < CONST_COPIED_PARAMS; ++i) {
    tmp += excess_params[i];
  }

  *result = tmp;
}

// Kernel with 32,764 byte kernel parameter limit
__global__ void kernelLargeParam(__grid_constant__ const param_large_t p, int *result) {
  int tmp = 0;

  // accumulate kernel parameters
  for (int i = 0; i < TOTAL_PARAMS; ++i) {
    tmp += p.param[i];
  }

  *result = tmp;
}

static void report_time(std::chrono::time_point<std::chrono::steady_clock> start,
                        std::chrono::time_point<std::chrono::steady_clock> end,
                        int iters) {
  auto usecs = duration_cast<duration<float,
                                      microseconds::period>>(end - start);
  cout << usecs.count() / iters << endl;
}

int main() {
  int rc;
  hipFree(0);

  param_t p;
  param_large_t p_large;

  // pageable host memory that holds excess constants passed via constant memory
  int *copied_params = (int *)malloc(CONST_COPIED_PARAMS * sizeof(int));
  assert(copied_params);

  // storage for computed result
  int *d_result;
  int h_result;
  checkCudaErrors(hipMalloc(&d_result, sizeof(int)));

  int expected_result = 0;

  // fill in data for validation
  for (int i = 0; i < KERNEL_PARAM_LIMIT; ++i) {
    p.param[i] = (i & 0xFF);
  }
  for (int i = KERNEL_PARAM_LIMIT; i < TOTAL_PARAMS; ++i) {
    copied_params[i - KERNEL_PARAM_LIMIT] = (i & 0xFF);
  }
  for (int i = 0; i < TOTAL_PARAMS; ++i) {
    p_large.param[i] = (i & 0xFF);
    expected_result += (i & 0xFF);
  }

  // warmup, verify correctness
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(excess_params), copied_params, CONST_COPIED_PARAMS * sizeof(int), 0, hipMemcpyHostToDevice));
  kernelDefault<<<1,1>>>(p, d_result);
  checkCudaErrors(hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipDeviceSynchronize());
  if(h_result != expected_result) {
    std::cout << "Test failed" << std::endl;
	 rc=-1;
	 goto Exit;    
  }

  kernelLargeParam<<<1,1>>>(p_large, d_result);
  checkCudaErrors(hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipDeviceSynchronize());
  if(h_result != expected_result) {
    std::cout << "Test failed" << std::endl;
	 rc=-1;
	 goto Exit;    
  }

  // benchmark default kernel parameter limit
  {
    auto start = steady_clock::now();
    for (int i = 0; i < TEST_ITERATIONS; ++i) {
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(excess_params), copied_params, CONST_COPIED_PARAMS * sizeof(int), 0, hipMemcpyHostToDevice));
      kernelDefault<<<1, 1>>>(p, d_result);
    }
    checkCudaErrors(hipDeviceSynchronize());
    auto end = steady_clock::now();
    std::cout << "Kernel 4KB parameter limit - time (us):";
    report_time(start, end, TEST_ITERATIONS);

    // benchmark large kernel parameter limit
    start = steady_clock::now();
    for (int i = 0; i < TEST_ITERATIONS; ++i) {
      kernelLargeParam<<<1, 1>>>(p_large, d_result);
    }  
    checkCudaErrors(hipDeviceSynchronize());
    end = steady_clock::now();
    std::cout << "Kernel 32,764 byte parameter limit - time (us):";
    report_time(start, end, TEST_ITERATIONS);
  }
  std::cout << "Test passed!" << std::endl;
  rc=0;
Exit:
  // cleanup
  hipFree(d_result);
  free(copied_params);
  return rc;
}
